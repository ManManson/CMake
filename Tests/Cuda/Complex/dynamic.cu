
#include <string>
#include <hip/hip_runtime.h>

int dynamic_base_func(int);

int __host__ cuda_dynamic_host_func(int x)
{
  return dynamic_base_func(x);
}

static
__global__
void DetermineIfValidCudaDevice()
{
}

void cuda_dynamic_lib_func(std::string& contents )
{
  DetermineIfValidCudaDevice <<<1,1>>> ();
  if(hipSuccess == hipGetLastError())
    {
    contents = "ran a cuda kernel";
    }
  else
    {
    contents = "cant run a cuda kernel";
    }
}
